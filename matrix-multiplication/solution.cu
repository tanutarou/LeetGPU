#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float *A, const float *B, float *C, int M, int N, int K)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < K && y < M)
    {
        float sum = 0.0;
        for (int i = 0; i < N; i++)
        {
            sum += A[y * N + i] * B[i * K + x];
        }
        C[y * K + x] = sum;
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float *A, const float *B, float *C, int M, int N, int K)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
